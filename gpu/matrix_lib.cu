#include "hip/hip_runtime.h"
#include "matrix_lib.h"
#include <stddef.h>
#include <stdio.h>
#include <unistd.h>

// global variables (consider making them constants)
size_t NUM_THREADS = 256; // You can adjust the number of threads per block
size_t NUM_BLOCKS = 256;  // You can adjust the number of blocks
size_t MAX_MEMORY = 1024 * 1024 * 1024; // You can adjust the maximum memory size in bytes (1 GB in this example)

int set_grid_size(size_t num_threads, size_t num_blocks, size_t max_memory) {
    // Check if the input values are valid
    if (num_threads == 0 || num_blocks == 0 || max_memory == 0) {
        return 0;
    }

    // Update the global variables
    NUM_THREADS = num_threads;
    NUM_BLOCKS = num_blocks;
    MAX_MEMORY = max_memory;

    return 1;
}

size_t get_array_size(Matrix *matrix) {
    return (size_t)matrix->width * (size_t) matrix->height;
}

void checkCudaError(hipError_t error) {
    if (error != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(error));
    }
}

__global__ void scalarMatrixMultKernel(float scalar_value, float* d_rows, size_t copy_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < copy_size; i += blockDim.x * gridDim.x) {
        // printf("index = %d\n", i);
        d_rows[i] *= scalar_value;
        // printf("value on index = %.2f\n", d_rows[i]);
    }
}

int scalar_matrix_mult(float scalar_value, Matrix *matrix) {
    hipError_t error;

    if (matrix->width > 0 && matrix->height > 0 && matrix->h_rows != NULL && matrix->d_rows != NULL) {
        const size_t array_size = get_array_size(matrix);
        size_t count = 0;
        printf("Array size = %ld\n", array_size);
        sleep(1);
        size_t remaining, copy_size;

        for (size_t i = 0; i < array_size; i += MAX_MEMORY) {
            printf("Array size = %ld\n", array_size);
            remaining = array_size - i;
            copy_size = (remaining < MAX_MEMORY) ? remaining : MAX_MEMORY;
            printf("Array size = %ld\n", array_size);
            // Copy data from host to device
            error = hipMemcpy(matrix->d_rows + i, matrix->h_rows + i, copy_size * sizeof(float), hipMemcpyHostToDevice);
            checkCudaError(error);

            // Launch the kernel
            scalarMatrixMultKernel<<<NUM_BLOCKS, NUM_THREADS>>>(scalar_value, matrix->d_rows + i, copy_size);

            // Copy the result back from device to host
            error = hipMemcpy(matrix->h_rows + i, matrix->d_rows + i, copy_size * sizeof(float), hipMemcpyDeviceToHost);
            checkCudaError(error);
            printf("%ld - ", MAX_MEMORY);
            printf("%ld\n", i);
            ++count;
            printf("Count = %ld\n", count);
            printf("Array Size = %ld\n", array_size);
        }

        // Wait for all threads to finish
        // hipDeviceSynchronize();

        // Free device memory
        printf("Vasco da Gama\n");
        error = hipFree(matrix->d_rows);
        checkCudaError(error);
    
        return 1;
    }

    return 0;
}
