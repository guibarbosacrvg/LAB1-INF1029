#include "hip/hip_runtime.h"
#include "matrix_lib.h"
#include <stddef.h>
#include <stdio.h>

// global variables
size_t NUM_THREADS = 1;
size_t NUM_BLOCKS = 1;
size_t MAX_MEMORY = 1;

int set_grid_size(size_t num_threads, size_t num_blocks, size_t max_memory){
    if(num_threads == 0 || num_blocks == 0 || max_memory == 0){
        return 0;
    }
    NUM_THREADS = num_threads;
    NUM_BLOCKS = num_blocks;
    MAX_MEMORY = max_memory;
    return 1;
}

size_t get_array_size(Matrix *matrix){
    return sizeof(float) * (size_t) matrix->width * matrix->height;
}

void checkCudaError(hipError_t error){
    if(error != hipSuccess){
        printf("Error: %s\n", hipGetErrorString(error));
    }
}

__global__ void scalarMatrixMultKernel(float scalar_value, float* d_rows, size_t array_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < array_size; i += blockDim.x * gridDim.x) {
        d_rows[i] *= scalar_value;
    }
}


int scalar_matrix_mult(float scalar_value, Matrix *matrix){
    hipError_t error;
    if(matrix->width > 0 && matrix->height > 0 && matrix->h_rows != NULL && matrix->d_rows != NULL){    
        size_t array_size = get_array_size(matrix);
        size_t remaining = array_size;
        size_t copy_size;
        float* d_scalar_value;
        error = hipMalloc((void**)&d_scalar_value, sizeof(float));
        checkCudaError(error);
        error = hipMemcpy(d_scalar_value, &scalar_value, sizeof(float), hipMemcpyHostToDevice);
        checkCudaError(error);
        for(size_t i = 0; i < array_size; i += MAX_MEMORY){
            remaining --;
            copy_size = (remaining < MAX_MEMORY) ? remaining : MAX_MEMORY;
            error = hipMemcpy(matrix->d_rows + i, matrix->h_rows + i, copy_size * sizeof(float), hipMemcpyHostToDevice);
            checkCudaError(error);
            scalarMatrixMultKernel<<<NUM_BLOCKS, NUM_THREADS>>>(scalar_value, matrix->d_rows + i, copy_size);
            error = hipMemcpy(matrix->h_rows + i, matrix->d_rows + i, copy_size * sizeof(float), hipMemcpyDeviceToHost);
            checkCudaError(error);
            error = hipFree(matrix->d_rows + i);
            checkCudaError(error);
        }
        hipDeviceSynchronize();
        return 1;
    }
    return 0;
}

